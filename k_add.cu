
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add_kernel(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    // on Host
    int a, b, c;
    // copy on Device
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);
    // allocate memory on device
    //use a pointer to address to be populated
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);


    // initialize a, b
    a = 4;
    b = 2;
    // copy memory from host to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // luanch the kernel to compute
    add_kernel<<<1, 1>>>(d_a, d_b, d_c);

    // since the result we need still be stored at device
    // so we have to copy it to host memory
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("res: %d\n", c);

    // all are done, so we can free all the memory we have allocated
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}